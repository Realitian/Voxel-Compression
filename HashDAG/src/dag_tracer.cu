#include "hip/hip_runtime.h"
#include "dag_tracer.h"
#include "cuda_error_check.h"
#include "memory.h"
#include "tracer.h"
#include "dags/basic_dag/basic_dag.h"
#include "dags/hash_dag/hash_dag.h"
#include "dags/hash_dag/hash_dag_colors.h"

DAGTracer::DAGTracer(bool headLess)
	: headLess(headLess)
{
    if (headLess)
    {
        const auto setupArray = [](auto& array, auto& buffer, auto x, auto y, auto z, auto w)
        {
            hipChannelFormatDesc desc = hipCreateChannelDesc(x, y, z, w, hipChannelFormatKindUnsigned);
            CUDA_CHECKED_CALL hipMallocArray(&array, &desc, imageWidth, imageHeight, hipArraySurfaceLoadStore);
            buffer.create_surface(array);
        };

        setupArray(pathArray, pathsBuffer, 32, 32, 32, 32);
        setupArray(colorsArray, colorsBuffer, 8, 8, 8, 8);
    }
    else
    {
        const auto setupImage = [](auto& buffer, auto& image, GLint formatA, GLenum formatB, GLenum formatC)
        {
            glGenTextures(1, &image);
            glBindTexture(GL_TEXTURE_2D, image);
            glTexImage2D(GL_TEXTURE_2D, 0, formatA, (int32)imageWidth, (int32)imageHeight, 0, formatB, formatC, nullptr);
            glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
            glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
            glBindTexture(GL_TEXTURE_2D, 0);
            buffer.register_resource(image);
        };

        setupImage(pathsBuffer, pathsImage, GL_RGBA32UI, GL_RGBA_INTEGER, GL_UNSIGNED_INT);
        setupImage(colorsBuffer, colorsImage, GL_RGBA, GL_RGBA, GL_UNSIGNED_BYTE);

        pathCache = Memory::malloc<uint3>("path cache", sizeof(uint3), EMemoryType::GPU_Managed);
    }

	hipEventCreate(&eventBeg);
	hipEventCreate(&eventEnd);
}


DAGTracer::~DAGTracer()
{
    if (headLess)
    {
		pathsBuffer.destroy_surface();
		colorsBuffer.destroy_surface();
		hipFreeArray(pathArray);
		hipFreeArray(colorsArray);
    }
    else
    {
        pathsBuffer.unregister_resource();
        colorsBuffer.unregister_resource();
        glDeleteTextures(1, &pathsImage);
        glDeleteTextures(1, &colorsImage);

        Memory::free(pathCache);
    }

	hipEventDestroy(eventBeg);
	hipEventDestroy(eventEnd);
}

inline Tracer::TracePathsParams get_trace_params(
	const CameraView& camera, 
	uint32 levels,
	const DAGInfo& dagInfo)
{
	const double3 position  = make_double3(camera.position);
	const double3 direction = make_double3(camera.forward());
	const double3 up        = make_double3(camera.up());
	const double3 right     = make_double3(camera.right());

	const double3 boundsMin = make_double3(dagInfo.boundsAABBMin);
	const double3 boundsMax = make_double3(dagInfo.boundsAABBMax);

	const double fov = camera.fov / 2.0 * (double(M_PI) / 180.);
	const double aspect_ratio = double(imageWidth) / double(imageHeight);
	
	const double3 X = right     * sin(fov) * aspect_ratio;
	const double3 Y = up        * sin(fov);
	const double3 Z = direction * cos(fov);

	const double3 bottomLeft  = position + Z - Y - X;
	const double3 bottomRight = position + Z - Y + X;
	const double3 topLeft     = position + Z + Y - X;

	const double3 translation = -boundsMin;
	const double3 scale = make_double3(double(1 << levels)) / (boundsMax - boundsMin);

	const double3 finalPosition    = (position    + translation) * scale;
	const double3 finalBottomLeft  = (bottomLeft  + translation) * scale;
	const double3 finalTopLeft     = (topLeft     + translation) * scale;
	const double3 finalBottomRight = (bottomRight + translation) * scale;
	const double3 dx = (finalBottomRight - finalBottomLeft) * (1.0 / imageWidth);
	const double3 dy = (finalTopLeft     - finalBottomLeft) * (1.0 / imageHeight);

	Tracer::TracePathsParams params;

	params.cameraPosition = finalPosition;
	params.rayMin = finalBottomLeft;
	params.rayDDx = dx;
	params.rayDDy = dy;

	return params;
}


template<typename TDAG>
float DAGTracer::resolve_paths(const CameraView& camera, const DAGInfo& dagInfo, const TDAG& dag)
{
	PROFILE_FUNCTION();
	
	const dim3 block_dim = dim3(4, 64);
	const dim3 grid_dim = dim3(imageWidth / block_dim.x + 1, imageHeight / block_dim.y + 1);

    if (!headLess) pathsBuffer.map_surface();
	auto traceParams = get_trace_params(camera, dag.levels, dagInfo);
	traceParams.pathsSurface = pathsBuffer.cudaSurface;

    CUDA_CHECK_ERROR();

	hipEventRecord(eventBeg);
	Tracer::trace_paths <<<grid_dim, block_dim>>> (traceParams, dag);
	hipEventRecord(eventEnd);
	hipEventSynchronize(eventEnd);

	CUDA_CHECK_ERROR();

	float elapsed;
	hipEventElapsedTime(&elapsed, eventBeg, eventEnd);
	CUDA_CHECK_ERROR();
	if (!headLess) pathsBuffer.unmap_surface();

	return elapsed;
}

template<typename TDAG, typename TDAGColors>
float DAGTracer::resolve_colors(const TDAG& dag, const TDAGColors& colors, EDebugColors debugColors, uint32 debugColorsIndexLevel, ToolInfo toolInfo)
{
	PROFILE_FUNCTION();
	
    colors.check_ready_for_rt();

	const dim3 block_dim = dim3(4, 64);
    const dim3 grid_dim = dim3(imageWidth / block_dim.x + 1, imageHeight / block_dim.y + 1);

	if (!headLess) pathsBuffer.map_surface();
	if (!headLess) colorsBuffer.map_surface();
	Tracer::TraceColorsParams traceParams;
	traceParams.debugColors = debugColors;
	traceParams.debugColorsIndexLevel = debugColorsIndexLevel;
	traceParams.toolInfo = toolInfo;
	traceParams.pathsSurface = pathsBuffer.cudaSurface;
	traceParams.colorsSurface = colorsBuffer.cudaSurface;

    CUDA_CHECK_ERROR();

	hipEventRecord(eventBeg);
	Tracer::trace_colors<<<grid_dim, block_dim>>>(traceParams, dag, colors);
	hipEventRecord(eventEnd);
	hipEventSynchronize(eventEnd);

	float elapsed;
	hipEventElapsedTime(&elapsed, eventBeg, eventEnd);
	CUDA_CHECK_ERROR();

	if (!headLess) pathsBuffer.unmap_surface();
	if (!headLess) colorsBuffer.unmap_surface();

	return elapsed;
}

template<typename TDAG>
float DAGTracer::resolve_shadows(const CameraView& camera, const DAGInfo& dagInfo, const TDAG& dag, float shadowBias, float fogDensity)
{
	PROFILE_FUNCTION();
	
    const dim3 block_dim = dim3(4, 64);
    const dim3 grid_dim = dim3(imageWidth / block_dim.x + 1, imageHeight / block_dim.y + 1);

    if (!headLess) pathsBuffer.map_surface();
	if (!headLess) colorsBuffer.map_surface();

	const auto pathParams = get_trace_params(camera, dag.levels, dagInfo);
    Tracer::TraceShadowsParams traceParams{
            pathParams.cameraPosition,
            pathParams.rayMin,
            pathParams.rayDDx,
            pathParams.rayDDy,
            shadowBias,
            fogDensity,
            pathsBuffer.cudaSurface,
            colorsBuffer.cudaSurface
    };

    CUDA_CHECK_ERROR();

	hipEventRecord(eventBeg);
	Tracer::trace_shadows <<<grid_dim, block_dim>>> (traceParams, dag);
	hipEventRecord(eventEnd);
	hipEventSynchronize(eventEnd);

	float elapsed;
	hipEventElapsedTime(&elapsed, eventBeg, eventEnd);
	CUDA_CHECK_ERROR();

	if (!headLess) pathsBuffer.unmap_surface();
	if (!headLess) colorsBuffer.unmap_surface();

	return elapsed;
}

template float DAGTracer::resolve_paths<BasicDAG>(const CameraView&, const DAGInfo&, const BasicDAG&);
template float DAGTracer::resolve_paths<HashDAG >(const CameraView&, const DAGInfo&, const HashDAG &);

template float DAGTracer::resolve_shadows<BasicDAG>(const CameraView&, const DAGInfo&, const BasicDAG&, float, float);
template float DAGTracer::resolve_shadows<HashDAG >(const CameraView&, const DAGInfo&, const HashDAG &, float, float);

#define COLORS_IMPL(Dag, Colors)\
template float DAGTracer::resolve_colors<Dag, Colors>(const Dag&, const Colors&, EDebugColors, uint32, ToolInfo);

COLORS_IMPL(BasicDAG, BasicDAGUncompressedColors)
COLORS_IMPL(BasicDAG, BasicDAGCompressedColors)
COLORS_IMPL(BasicDAG, BasicDAGColorErrors)
COLORS_IMPL(HashDAG, HashDAGColors)

__global__ void read_path(uint32 x, uint32 y, hipSurfaceObject_t surface, uint3* output)
{
	*output = make_uint3(surf2Dread<uint4>(surface, x * sizeof(uint4), y));
}

uint3 DAGTracer::get_path(uint32 posX, uint32 posY)
{
	PROFILE_FUNCTION();
	
    if (headLess) return {};

	check(posX < imageWidth);
    check(posY < imageHeight);

    pathsBuffer.map_surface();
    CUDA_CHECK_ERROR();
	read_path<<<1,1>>>(posX, posY, pathsBuffer.cudaSurface, pathCache);
	CUDA_CHECK_ERROR();
    if (!headLess) pathsBuffer.unmap_surface();

    return *pathCache;
}
